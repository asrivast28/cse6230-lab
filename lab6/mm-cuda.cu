/**
 *  \file mm-cuda.cu
 *  \brief CUBLAS-based implementation of the local matrix multiply.
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

static int isInit__ = 0; /*!< Set to '1' when handle is initialized, '-1' on error */
static hipblasHandle_t handle__;

extern void sgemm_ (const char* opA, const char* opB,
		    const int* M, const int* N, const int* K,
		    const float* alpha, const float* A, const int* lda,
		    const float* B, const int* ldb,
		    const float* beta, float* C, const int* ldc);

static void mm_init (void);
static void assert_status (hipblasStatus_t stat);

/* ======================================================================
 * Modify the three routines in this section (mm_local, mm_create, and
 * mm_free) to improve the efficiency of the GPU implementation.
 */

extern "C"
void
mm_local (const int m, const int n, const int k,
	  const float* A_cpu_pinned, const int lda,
	  const float* B_cpu_pinned, const int ldb,
	  float* C_cpu_pinned, const int ldc)
{
  mm_init ();
  assert (A_cpu_pinned || m <= 0 || k <= 0); assert (lda >= m);
  assert (B_cpu_pinned || k <= 0 || n <= 0); assert (ldb >= k);
  assert (C_cpu_pinned || m <= 0 || n <= 0); assert (ldc >= m);

  float* A_gpu;
  hipMalloc ((void **)&A_gpu, m * k * sizeof (float)); assert (A_gpu);
  float* B_gpu;
  hipMalloc ((void **)&B_gpu, k * n * sizeof (float)); assert (B_gpu);
  float* C_gpu;
  hipMalloc ((void **)&C_gpu, m * n * sizeof (float)); assert (C_gpu);
  hipMemcpy (A_gpu, A_cpu_pinned, m * k * sizeof (float), hipMemcpyHostToDevice);
  hipMemcpy (B_gpu, B_cpu_pinned, k * n * sizeof (float), hipMemcpyHostToDevice);
  hipMemcpy (C_gpu, C_cpu_pinned, m * n * sizeof (float), hipMemcpyHostToDevice);

  const float ONE = 1.0;
  hipblasStatus_t stat = hipblasSgemm (handle__, HIPBLAS_OP_N, HIPBLAS_OP_N,
				     m, n, k,
				     &ONE, A_gpu, lda, B_gpu, ldb,
				     &ONE, C_gpu, ldc);
  assert_status (stat);

  hipMemcpy (C_cpu_pinned, C_gpu, m * n * sizeof (float), hipMemcpyDeviceToHost);
  hipFree (C_gpu);
  hipFree (B_gpu);
  hipFree (A_gpu);
}

extern "C"
float *
mm_create (int m, int n)
{
  float* A_cpu_pinned = NULL;
  hipHostAlloc ((void **)&A_cpu_pinned, m * n * sizeof(float), hipHostMallocMapped | hipHostMallocPortable);
  assert (A_cpu_pinned);
  return A_cpu_pinned;
}

extern "C"
void
mm_free (float* A_cpu_pinned)
{
  if (A_cpu_pinned) hipHostFree (A_cpu_pinned);
}

/* ====================================================================== */

static
void
mm_init (void)
{
  if (!isInit__) {
    hipblasStatus_t stat = hipblasCreate (&handle__);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "*** CUBLAS initialization failure!\n");
      exit (-1);
    }
    isInit__ = 1;
  }
}

static
void
assert_status (hipblasStatus_t stat)
{
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    switch (stat) {
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      fprintf (stderr, "CUBLAS_STATUS_NOT_INITIALIZED\n");
      break;
    case HIPBLAS_STATUS_INVALID_VALUE:
      fprintf (stderr, "CUBLAS_STATUS_INVALID_VALUE\n");
      break;
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      fprintf (stderr, "CUBLAS_STATUS_ARCH_MISMATCH\n");
      break;
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      fprintf (stderr, "CUBLAS_STATUS_EXECUTION_FAILED\n");
      break;
    default:
      fprintf (stderr, "(unknown error)\n");
      break;
    }
  }
}

/* eof */
