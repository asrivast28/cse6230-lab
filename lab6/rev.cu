#include "hip/hip_runtime.h"
/**
 *  \file rev.cu
 *  \brief CUDA unified virtual addressing benchmark example (reverse
 *  a list).
 */

#include <assert.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>

#include "timer.c"

#define MINTIME 2.0 /*!< Minimum running time, in seconds */
#define MINTRIALS 3 /*!< Minimum number of timing trials, in seconds */
#define MAXVAL 1000 /*!< Maximum array value */

#if CUDART_VERSION < 2020
#  error "*** Must have a CUDA 2.2 or greater. ***"
#endif

/** Initialize A[i] = i % MAXVAL */
static void initArray (size_t n, int* A);

/** Checks that A[i] == i % MAXVAL */
static void verifyArray (size_t n, int* A);

/* ================================================== */

/** 
 *  Copies data from A_cpu (on the CPU) to A_gpu (on the GPU),
 *  reverses the data elements on the GPU, and copies the data back.
 */
__global__
void reverseArray (long n, int* A_gpu)
{
  long i = blockIdx.x * blockDim.x + threadIdx.x;
  const long n_half = n >> 1; /* floor (n / 2) */
  if (i < n_half) {
    const int i_pair = n - i - 1;
    int a = A_gpu[i];
    int b = A_gpu[i_pair];
    A_gpu[i] = b;
    A_gpu[i_pair] = a;
  }
}

/* ================================================== */

/** Benchmarks the reversal operation on unpinned memory. */
static
long double
benchmarkReverseWithCopies (size_t n, int* A_cpu)
{
  int* A_gpu = NULL;
  hipMalloc ((void **)&A_gpu, n * sizeof (int)); assert (A_gpu);

  /* Do one test run */
  fprintf (stderr, "benchmarkReverseWithCopies: Testing...\n");
  const int BLOCKSIZE = 1024;
  const int NUMBLOCKS = (((n+1)/2) + BLOCKSIZE - 1) / BLOCKSIZE;
  initArray (n, A_cpu);
  hipMemcpy (A_gpu, A_cpu, n * sizeof (int), hipMemcpyDefault);
  reverseArray <<<NUMBLOCKS, BLOCKSIZE>>> (n, A_gpu);
  hipMemcpy (A_cpu, A_gpu, n * sizeof (int), hipMemcpyDefault);
  verifyArray (n, A_cpu);
  fprintf (stderr, "==> Passed!\n\n");

  /* Timing runs */
  fprintf (stderr, "benchmarkReverseWithCopies: Timing...\n");
  long double t_elapsed = 0;
  size_t trials = 0;
  stopwatch_init ();
  struct stopwatch_t* timer = stopwatch_create ();
  stopwatch_start (timer);
  while (trials < MINTRIALS || t_elapsed < MINTIME) {
    hipMemcpy (A_gpu, A_cpu, n * sizeof (int), hipMemcpyDefault);
    reverseArray <<<NUMBLOCKS, BLOCKSIZE>>> (n, A_gpu);
    hipMemcpy (A_cpu, A_gpu, n * sizeof (int), hipMemcpyDefault);
    hipDeviceSynchronize ();
    ++trials;
    t_elapsed = stopwatch_elapsed (timer);
  }
  stopwatch_destroy (timer);
  fprintf (stderr, "==> %lu trials took %Lg seconds.\n", trials, t_elapsed);
  hipFree (A_gpu);
  return t_elapsed / trials;
}

/* ================================================== */

/** Benchmarks the reversal operation on pinned memory. */
static
long double
benchmarkReverseWithoutCopies (size_t n, int* A_cpu_pinned)
{
  /* Do one test run */
  fprintf (stderr, "benchmarkReverseWithoutCopies: Testing...\n");
  const int BLOCKSIZE = 1024;
  const int NUMBLOCKS = (((n+1)/2) + BLOCKSIZE - 1) / BLOCKSIZE;
  initArray (n, A_cpu_pinned);
  reverseArray <<<NUMBLOCKS, BLOCKSIZE>>> (n, A_cpu_pinned);
  hipDeviceSynchronize ();
  verifyArray (n, A_cpu_pinned);
  fprintf (stderr, "==> Passed!\n\n");

  /* Timing runs */
  fprintf (stderr, "benchmarkReverseWithoutCopies: Timing...\n");
  long double t_elapsed = 0;
  size_t trials = 0;
  stopwatch_init ();
  struct stopwatch_t* timer = stopwatch_create ();
  stopwatch_start (timer);
  while (trials < MINTRIALS || t_elapsed < MINTIME) {
    reverseArray <<<NUMBLOCKS, BLOCKSIZE>>> (n, A_cpu_pinned);
    hipDeviceSynchronize ();
    ++trials;
    t_elapsed = stopwatch_elapsed (timer);
  }
  stopwatch_destroy (timer);
  fprintf (stderr, "==> %lu trials took %Lg seconds.\n", trials, t_elapsed);
  return t_elapsed / trials;
}

/* ================================================== */

#define TARGET(i) ((int)((i) % MAXVAL))

static
void
initArray (size_t n, int* A)
{
  for (size_t i = 0; i < n; ++i) {
    const int target = TARGET (i);
    A[i] = target;
  }
}

static
void
verifyArray (size_t n, int* A)
{
  for (size_t i = 0; i < n; ++i) {
    const int target = TARGET (n - i - 1);
    if (A[i] != target) {
      fprintf (stderr, "*** ERROR: Element A[%lu] == %d != %d! ***\n",
	       (unsigned long)i, A[i], target);
      assert (0);
    }
  }
}

/* ================================================== */

int
main (int argc, char* argv[])
{
  if (argc < 2) {
    fprintf (stderr, "usage: %s <n>\n", argv[0]);
    return -1;
  }

  long n_raw = atol (argv[1]);
  assert (n_raw > 0);
  const size_t n = (size_t)n_raw;
  const size_t n_bytes = n * sizeof (int);

  fprintf (stderr, "n = %lu (~ %.1f MiB)\n", n, (double)n_bytes/1024/1024);

  fprintf (stderr, "Test 1: Using 'malloc' on CPU...\n");
  int* A_cpu = (int *)malloc (n_bytes);
  assert (A_cpu);
  long double t_baseline = benchmarkReverseWithCopies (n, A_cpu);
  printf ("==> Reversal with explicit copies: %Lg seconds (%Lg effective GB/s)\n\n",
	  t_baseline, (long double)2e-9 * n_bytes / t_baseline);
  free (A_cpu);

  fprintf (stderr, "Test 2: Using pinned hipHostAlloc...\n");
  int* A_cpu_pinned = NULL;
  hipHostAlloc ((void **)&A_cpu_pinned, n_bytes, hipHostMallocMapped | hipHostMallocPortable);
  assert (A_cpu_pinned);
  long double t_pinned = benchmarkReverseWithoutCopies (n, A_cpu_pinned);
  printf ("==> Reversal without explicit copies: %Lg seconds (%Lg effective GB/s)\n\n",
	  t_pinned, (long double)2e-9 * n_bytes / t_pinned);
  hipHostFree (A_cpu_pinned);

  return 0;
}

/* eof */
