#include "hip/hip_runtime.h"
/**
 *  \file mgpu-dma.cu
 *  \brief Demo of how to use multiple GPUs.
 */

#include <assert.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>

#include "timer.c"

#define MINTIME 1 /*!< Minimum running time, in seconds */
#define MINTRIALS 3 /*!< Minimum number of timing trials, in seconds */
#define MAXVAL 1000 /*!< Maximum array value */
#define NUMEXP 10 /*!< Number of experimental trials */

#if CUDART_VERSION < 2020
#  error "*** Must have a CUDA 2.2 or greater. ***"
#endif

/**
 *  Performs a ping-pong through an intermediate buffer.
 */
static
void
pingpongBuffered (size_t n, int* A, int* B, int* tmpbuf)
{
  hipMemcpy (tmpbuf, A, n * sizeof (int), hipMemcpyDefault);
  hipMemcpy (B, tmpbuf, n * sizeof (int), hipMemcpyDefault);
  hipMemcpy (tmpbuf, B, n * sizeof (int), hipMemcpyDefault);
  hipMemcpy (A, tmpbuf, n * sizeof (int), hipMemcpyDefault);
  hipDeviceSynchronize ();
}

/**
 *  Performs a "ping-pong," which is a round-trip pair of cudaMemcpys,
 *  the first from A to B and the second immediately returning from B
 *  to A.
 */
static
void
pingpongDirect (size_t n, int* A, int* B)
{
  hipMemcpy (B, A, n * sizeof (int), hipMemcpyDefault);
  hipMemcpy (A, B, n * sizeof (int), hipMemcpyDefault);
  hipDeviceSynchronize ();
}

/**
 *  Performs a bunch of pingpongs (see above), and returns the average
 *  running time for each. "A bunch" means at least MINTRIALS
 *  pingpongs or MINTIME seconds, whichever occurs last. See also:
 *  pingpong().
 */
static long double benchmarkCopy (size_t n, int* A, int* B);

/**
 *  Performs a bunch of buffered ping-pongs, and returns the average
 *  running time for each. "A bunch" means at least MINTRIALS
 *  pingpongs or MINTIME seconds, whichever occurs last. See also:
 *  pingpongBuffered().
 */
static long double benchmarkBufferedCopy (size_t n, int* A, int* B, int* buf);

int
main (int argc, char* argv[])
{
  if (argc < 2) {
    fprintf (stderr, "usage: %s <n>\n", argv[0]);
    return -1;
  }

  long n_raw = atol (argv[1]);
  assert (n_raw > 0);
  const size_t n = (size_t)n_raw;
  const size_t n_bytes = n * sizeof (int);

  fprintf (stderr, "n = %lu (~ %.1f MiB)\n\n", n, (double)n_bytes/1024/1024);

  /* Timer setup */
  stopwatch_init ();

  /* Determine how many GPUs there are. Need at least two for this demo. */
  int gpu_count = 0;
  hipGetDeviceCount (&gpu_count);
  fprintf (stderr, "Detected %d GPUs.\n", gpu_count);
  assert (gpu_count >= 2);

  /* Allocate arrays on each of two GPUs */
  int* A_gpu[2] = {NULL, NULL};
  hipSetDevice (0);
  hipMalloc ((void **)&(A_gpu[0]), n_bytes); assert (A_gpu[0]);
  hipSetDevice (1);
  hipMalloc ((void **)&(A_gpu[1]), n_bytes); assert (A_gpu[1]);

  /* Test 1: Ping-pong, using a temporary CPU buffer */
  fprintf (stderr, "Test 1: Buffering GPU-to-GPU copy using CPU memory...\n");
  int* A_cpu = (int *)malloc (n_bytes); assert (A_cpu);

  pingpongBuffered (n, A_gpu[0], A_gpu[1], A_cpu); /* warm-up */

  long double t_baseline = benchmarkBufferedCopy (n, A_gpu[0], A_gpu[1], A_cpu);
  printf ("==> GPU-to-GPU copy, buffered through CPU memory: %Lg seconds (%Lg GB/s)\n\n",
	  t_baseline, (long double)2e-9 * n_bytes / t_baseline);
  free (A_cpu);

  fprintf (stderr, "Test 2: Direct GPU-to-GPU copy...\n");
  pingpongDirect (n, A_gpu[0], A_gpu[1]);

  long double t_dma = benchmarkCopy (n, A_gpu[0], A_gpu[1]);
  printf ("==> Direct GPU-to-GPU copy: %Lg seconds (%Lg GB/s)\n\n",
	  t_dma, (long double)2e-9 * n_bytes / t_dma);

  hipFree (A_gpu[0]);
  hipFree (A_gpu[1]);
  return 0;
}

static
long double
benchmarkCopy (size_t n, int* A, int* B)
{
  struct stopwatch_t* timer = stopwatch_create (); assert (timer);

  long double t_elapsed = 0;
  size_t trials = 0;
  stopwatch_start (timer);
  while (trials < MINTRIALS || t_elapsed < MINTIME) {
    pingpongDirect (n, A, B);
    ++trials;
    t_elapsed = stopwatch_elapsed (timer);
  }
  t_elapsed = stopwatch_stop (timer);
  stopwatch_destroy (timer);
  fprintf (stderr, "  [%lu trials]\n", (unsigned long)trials);
  return t_elapsed / trials;
}

static
long double
benchmarkBufferedCopy (size_t n, int* A, int* B, int* tmpbuf)
{
  struct stopwatch_t* timer = stopwatch_create (); assert (timer);

  long double t_elapsed = 0;
  size_t trials = 0;
  stopwatch_start (timer);
  while (trials < MINTRIALS || t_elapsed < MINTIME) {
    pingpongBuffered (n, A, B, tmpbuf);
    ++trials;
    t_elapsed = stopwatch_elapsed (timer);
  }
  t_elapsed = stopwatch_stop (timer);
  stopwatch_destroy (timer);
  fprintf (stderr, "  [%lu trials]\n", (unsigned long)trials);
  return t_elapsed / trials;
}

/* eof */
