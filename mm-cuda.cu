/**
 *  \file mm-cuda.cu
 *  \brief CUBLAS-based implementation of the local matrix multiply.
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

static int isInit__ = 0; /*!< Set to '1' when handle is initialized, '-1' on error */
static hipblasHandle_t handle__;

extern void sgemm_ (const char* opA, const char* opB,
		    const int* M, const int* N, const int* K,
		    const float* alpha, const float* A, const int* lda,
		    const float* B, const int* ldb,
		    const float* beta, float* C, const int* ldc);

static void mm_init (void);
static void assert_status (hipblasStatus_t stat);

/* ======================================================================
 * Modify the three routines in this section (mm_local, mm_create, and
 * mm_free) to improve the efficiency of the GPU implementation.
 */

extern "C"
void
mm_local (const int m, const int n, const int k,
	  const float* A, const int lda,
	  const float* B, const int ldb,
	  float* C, const int ldc)
{
  mm_init ();
  assert (A || m <= 0 || k <= 0); assert (lda >= m);
  assert (B || k <= 0 || n <= 0); assert (ldb >= k);
  assert (C || m <= 0 || n <= 0); assert (ldc >= m);

  float* A_gpu;
  hipMalloc ((void **)&A_gpu, m * k * sizeof (float)); assert (A_gpu);
  float* B_gpu;
  hipMalloc ((void **)&B_gpu, k * n * sizeof (float)); assert (B_gpu);
  float* C_gpu;
  hipMalloc ((void **)&C_gpu, m * n * sizeof (float)); assert (C_gpu);
  hipMemcpy (A_gpu, A, m * k * sizeof (float), hipMemcpyDefault);
  hipMemcpy (B_gpu, B, k * n * sizeof (float), hipMemcpyDefault);
  hipMemcpy (C_gpu, C, m * n * sizeof (float), hipMemcpyDefault);

  const float ONE = 1.0;
  hipblasStatus_t stat = hipblasSgemm (handle__, HIPBLAS_OP_N, HIPBLAS_OP_N,
				     m, n, k,
				     &ONE, A_gpu, lda, B_gpu, ldb,
				     &ONE, C_gpu, ldc);
  assert_status (stat);

  hipMemcpy (C, C_gpu, m * n * sizeof (float), hipMemcpyDefault);
  hipFree (C_gpu);
  hipFree (B_gpu);
  hipFree (A_gpu);
}

extern "C"
float *
mm_create (int m, int n)
{
  float* A = (float *)malloc (m * n * sizeof (float));
  assert (A);
  return A;
}

extern "C"
void
mm_free (float* A)
{
  if (A) free (A);
}

/* ====================================================================== */

static
void
mm_init (void)
{
  if (!isInit__) {
    hipblasStatus_t stat = hipblasCreate (&handle__);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf (stderr, "*** CUBLAS initialization failure!\n");
      exit (-1);
    }
    isInit__ = 1;
  }
}

static
void
assert_status (hipblasStatus_t stat)
{
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    switch (stat) {
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      fprintf (stderr, "CUBLAS_STATUS_NOT_INITIALIZED\n");
      break;
    case HIPBLAS_STATUS_INVALID_VALUE:
      fprintf (stderr, "CUBLAS_STATUS_INVALID_VALUE\n");
      break;
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      fprintf (stderr, "CUBLAS_STATUS_ARCH_MISMATCH\n");
      break;
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      fprintf (stderr, "CUBLAS_STATUS_EXECUTION_FAILED\n");
      break;
    default:
      fprintf (stderr, "(unknown error)\n");
      break;
    }
  }
}

/* eof */
