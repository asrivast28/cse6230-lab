#include "hip/hip_runtime.h"
// -*- mode:c++; tab-width:2; indent-tabs-mode:nil;  -*-
/**
 *  \file listrank-cuda.cu
 *
 *  \brief Implement the 'listrank-par.hh' interface using CUDA.
 */

#include <cassert>
#include <cstring>

#include <algorithm>
#include <iostream>

#include "listrank-par.hh"

#include "cuda_utils.h"

using namespace std;

// ============================================================
const char *
getImplName__par (void)
{
  return "CUDA";
}

// ============================================================

struct ParRankedList_t__
{
  size_t n;

  // Buffers on the host (i.e., CPU)
  const index_t* Next_host;
  rank_t* Rank_host;

  // Buffers on the device (i.e., GPU)
  index_t* Next_device;
  rank_t* Rank_device;
};

// ============================================================

ParRankedList_t *
setupRanks__par (size_t n, const index_t* Next)
{
  ParRankedList_t* L = new ParRankedList_t;
  assert (L);

  L->n = n;
  L->Next_host = Next;
  L->Rank_host = createRanksBuffer (n);

  // Create buffers on the GPU:
  CUDA_CHECK_ERROR (hipMalloc (&(L->Next_device), n * sizeof (index_t)));
  CUDA_CHECK_ERROR (hipMalloc (&(L->Rank_device), n * sizeof (rank_t)));

  // Copy CPU buffer contents to the GPU:
  CUDA_CHECK_ERROR (hipMemcpy (L->Next_device, L->Next_host,
                                n * sizeof (index_t),
                                hipMemcpyHostToDevice));
  CUDA_CHECK_ERROR (hipMemcpy (L->Rank_device, L->Rank_host,
                                n * sizeof (rank_t),
                                hipMemcpyHostToDevice));

  return L;
}

void releaseRanks__par (ParRankedList_t* L)
{
  if (L) {
    releaseRanksBuffer (L->Rank_host);

    // Free GPU buffers:
    CUDA_CHECK_ERROR (hipFree (L->Next_device));
    CUDA_CHECK_ERROR (hipFree (L->Rank_device));
  }
}

// ============================================================

const rank_t *
getRanks__par (const ParRankedList_t* L)
{
  // Copy GPU results back to the CPU:
  CUDA_CHECK_ERROR (hipMemcpy (L->Rank_host, L->Rank_device,
                                L->n * sizeof (rank_t),
                                hipMemcpyDeviceToHost));
  return L->Rank_host;
}

// ============================================================

/**
 *  Implements a GPU kernel to initialize the rank buffer. By analogy,
 *  see the first for loop in both the sequential and Cilk Plus
 *  implementations.
 */
__global__ void
computeListRanks__init (size_t n, const index_t* Next, rank_t* Rank)
{
  const unsigned int BID = blockIdx.y * gridDim.x + blockIdx.x; // block ID
  const unsigned int LID = threadIdx.x;  // local thread ID
  const int TPB = blockDim.x; // threads per block
  const unsigned int k = BID * TPB + LID;
  if (k < n)
    Rank[k] = (Next[k] == NIL) ? 0 : 1;
}

//#include "soln--cuda1.cu" // Instructor's solution: none for you!

void
computeListRanks__par (ParRankedList_t* L)
{
  assert (L != NULL);
  if (L->n == 0) return; // empty pool

  static const int MAX_THREADS_PER_BLOCK = 1024;
  static const int GB_X = 16;
  int blocks = (L->n + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
  dim3 GB (GB_X, (blocks + GB_X - 1) / GB_X, 1);
  dim3 TB (MAX_THREADS_PER_BLOCK, 1, 1);

  computeListRanks__init<<<GB, TB>>> (L->n, L->Next_device, L->Rank_device);

  //------------------------------------------------------------
  //
  // ... YOUR CODE GOES HERE ...
  //
  // (you may also modify the preceding code if you wish)
  //
  //#include "soln--cuda2.cu"  // Instructor's solution: none for you!
  //------------------------------------------------------------
}

// eof
