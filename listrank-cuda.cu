#include "hip/hip_runtime.h"
// -*- mode:c++; tab-width:2; indent-tabs-mode:nil;  -*-
/**
 *  \file listrank-cuda.cu
 *
 *  \brief Implement the 'listrank-par.hh' interface using CUDA.
 */

#include <cassert>
#include <cstring>

#include <algorithm>
#include <iostream>

#include "listrank-par.hh"

#include "cuda_utils.h"

#define hipDeviceScheduleBlockingSync 0x04


using namespace std;

// ============================================================
const char *
getImplName__par (void)
{
  return "CUDA";
}

// ============================================================

struct ParRankedList_t__
{
  size_t n;

  // Buffers on the host (i.e., CPU)
  const index_t* Next_host;
  rank_t* Rank_host;

  // Buffers on the device (i.e., GPU)
  index_t* Next_device[2];
  rank_t* Rank_device[2];
};

// ============================================================

ParRankedList_t *
setupRanks__par (size_t n, const index_t* Next)
{
  ParRankedList_t* L = new ParRankedList_t;
  assert (L);

  L->n = n;
  L->Next_host = Next;
  L->Rank_host = createRanksBuffer (n);

  // Create buffers on the GPU:
  for (size_t i = 0; i < 2; ++i) {
    CUDA_CHECK_ERROR (hipMalloc (&(L->Next_device[i]), n * sizeof (index_t)));
    CUDA_CHECK_ERROR (hipMalloc (&(L->Rank_device[i]), n * sizeof (rank_t)));
  }


  // Copy CPU buffer contents to the GPU:
  CUDA_CHECK_ERROR (hipMemcpy (L->Next_device[0], L->Next_host,
                                n * sizeof (index_t),
                                hipMemcpyHostToDevice));
  return L;
}

void releaseRanks__par (ParRankedList_t* L)
{
  if (L) {
    releaseRanksBuffer (L->Rank_host);

    // Free GPU buffers:
    for (size_t i = 0; i < 2; ++i) {
      CUDA_CHECK_ERROR (hipFree (L->Next_device[i]));
      CUDA_CHECK_ERROR (hipFree (L->Rank_device[i]));
    }
    L->Next_host = NULL;
    L->Rank_host = NULL;
  }
}

// ============================================================

const rank_t *
getRanks__par (const ParRankedList_t* L)
{
  // Copy GPU results back to the CPU:
  CUDA_CHECK_ERROR (hipMemcpy (L->Rank_host, L->Rank_device[0],
                                L->n * sizeof (rank_t),
                                hipMemcpyDeviceToHost));
  return L->Rank_host;
}

// ============================================================

/**
 *  Implements a GPU kernel to initialize the rank buffer. By analogy,
 *  see the first for loop in both the sequential and Cilk Plus
 *  implementations.
 */
__global__ void
initListRanks__init (const size_t n, rank_t* Rank, const index_t* Next)
{
  const unsigned int BID = blockIdx.y * gridDim.x + blockIdx.x; // block ID
  const unsigned int LID = threadIdx.x;  // local thread ID
  const int TPB = blockDim.x; // threads per block
  const unsigned int k = BID * TPB + LID;

  if (k < n)
    Rank[k] = (Next[k] == NIL) ? 0 : 1;
}

__global__ void
computeListRanks__init (ParRankedList_t L)
{
  const unsigned int BID = blockIdx.y * gridDim.x + blockIdx.x; // block ID
  const unsigned int LID = threadIdx.x;  // local thread ID
  const int TPB = blockDim.x; // threads per block
  const unsigned int k = BID * TPB + LID;

  if (k >= L.n) {
    return;
  }

  index_t* N_cur = L.Next_device[0];
  index_t* N_next = L.Next_device[1];

  rank_t* R_cur = L.Rank_device[0];
  rank_t* R_next = L.Rank_device[1];

  if (N_cur[k] != NIL) {
    R_next[k] = R_cur[k] + R_cur[N_cur[k]];
    N_next[k] = N_cur[N_cur[k]];
  }
  else {
    R_next[k] = R_cur[k];
    N_next[k] = NIL;
  }
}

void
computeListRanks__par (ParRankedList_t* L)
{
  assert (L != NULL);
  if (L->n == 0) return; // empty pool

  static const int MAX_THREADS_PER_BLOCK = 1024;
  static const int GB_X = 16;
  int blocks = (L->n + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
  dim3 GB (GB_X, (blocks + GB_X - 1) / GB_X, 1);
  dim3 TB (MAX_THREADS_PER_BLOCK, 1, 1);

  size_t maxIterations = static_cast<size_t>(ceil(log2(static_cast<double>(L->n)))); 

  initListRanks__init<<<GB, TB>>> (L->n, L->Rank_device[0], L->Next_device[0]);

  for (size_t it = 0; it < maxIterations; ++it) {
    computeListRanks__init<<<GB, TB>>> (*L);

    index_t* N_temp = L->Next_device[0];
    L->Next_device[0] = L->Next_device[1];
    L->Next_device[1] = N_temp;

    rank_t* R_temp = L->Rank_device[0];
    L->Rank_device[0] = L->Rank_device[1];
    L->Rank_device[1] = R_temp;
  }
  hipDeviceSynchronize();
}

// eof
